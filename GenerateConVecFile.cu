#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "mycurand.h"
#include "librairies.h"

#include "devFunctionProtos.h"
#include "devHostConstants.h"
#include "Matrix_Utils.cu"
#include "GenConProbDistDepMat.cu"

///////////////////////////////////////////////////////////////////    

void __cudaCheck(hipError_t err, const char* file, const int line);
#define cudaCheck(err) __cudaCheck (err, __FILE__, __LINE__)

void __cudaCheckLastError(const char* errorMessage, const char* file, const int line);
#define cudaCheckLastError(msg) __cudaCheckLastError (msg, __FILE__, __LINE__)

void __cudaCheck(hipError_t err, const char *file, const int line) {
  if( hipSuccess != err) {
    fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
      file, line, (int)err, hipGetErrorString( err ) );
    exit(-1);
  }
}

void __cudaCheckLastError(const char *errorMessage, const char *file, const int line) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
      file, line, errorMessage, (int)err, hipGetErrorString( err ) );
    exit(-1);
  }
}

///////////////////////////////////////////////////////////////////    

__host__ void nbNeurons(int N, int* &Nk) {
  
  cudaCheck(hipHostMalloc((void **)&Nk, nbpop * sizeof(int)));
  printf("Number of neurons : ") ;
  int i = 0;
  while(i<nbpop) {
    if(i==0)
      Nk[i] = N_NEURONS*popSize ;
    else
      Nk[i] = ( N_NEURONS * (100 - int( popSize * 100 ) ) / 100 ) / max( (nbpop-1), 1 ) ;
    
    printf("%d ", Nk[i]) ;
    ++i ;
  }
  printf("\n") ;
}

///////////////////////////////////////////////////////////////////    
 
__host__ void CptNeurons(int *Nk, int* &Cpt) {
  cudaCheck(hipHostMalloc((void **)&Cpt, nbpop * sizeof(int)));
  printf("Counter : ") ;
  for(int i=0;i<nbpop+1;i++) {
    Cpt[i] = 0 ;
    for(int j=0;j<i;j++) {
      Cpt[i] = Cpt[i] + Nk[j] ; 
    }
    printf("%d ", Cpt[i]) ;
  }
  printf("\n") ;
}

__global__ void initConVec(float *dev_conVec, int maxNeurons) {
  unsigned long int id = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned long int i;
  if(id < maxNeurons) 
    for(i = 0; i < N_NEURONS; i++) 
      dev_conVec[i + id * maxNeurons] = 0 ; 
}

__global__ void initPreFactor(float *dev_preFactor) {
  unsigned long int i;
  for(i = 0; i < 2 * N_NEURONS ; i++) 
    dev_preFactor[i] = 0 ; 
}

// __global__ void initIdPost(int *dev_IdPost) {
//   unsigned long int i;
//   for(i = 0; i < N_NEURONS ; i++) 
//     dev_IdPost[i] = 0 ; 
// }

// __global__ void initNbPost(int *dev_nbPost) {
//   unsigned long int i;
//   for(i = 0; i < N_NEURONS ; i++) 
//     dev_nbPost[i] = 0 ; 
// }

// __global__ void initNbPreS(int *dev_nbPreS) {
//   unsigned long int i;
//   for(i = 0; i < nbpop * nbpop ; i++) 
//     dev_nbPreS[i] = 0 ; 
// }

__global__ void setup_kernel(hiprandState *state, unsigned long long seed ) {
  unsigned long int id = threadIdx.x + blockIdx.x * blockDim.x;
  /* Each thread gets different seed, a different sequence number, no offset */
  if(id < N_NEURONS) 
    hiprand_init(seed * (id + 7), id, 0, &state[id]);
}

__device__ float randkernel(hiprandState *state, unsigned long int kNeuron) {
  /*RETURNS ONE SAMPLE FROM UNIFORM DISTRIBUTION*/
  /*  unsigned int id = (unsigned long int)threadIdx.x + blockIdx.x * blockDim.x;*/
  float randNumber= 0.0;
  if(kNeuron < N_NEURONS) {
    hiprandState localState = state[kNeuron]; /* state in global memory */
    randNumber = hiprand_uniform(&localState);
    state[kNeuron] = localState;
  }
  return randNumber;
}

__global__ void kernelGenConMat(hiprandState *state, float *dev_conVec, int lChunck, int maxNeurons, int* nbN) {

  /* indexing of matrix row + clm x N_NEURONS*/
  unsigned long id =  (unsigned long int)threadIdx.x + blockIdx.x * blockDim.x;
  unsigned long int kNeuron = id + lChunck * maxNeurons;
  unsigned long int i;
  
  if(id < maxNeurons & kNeuron < N_NEURONS) 
    for(i=0; i<N_NEURONS; i++) // j is row and id is clmn 
      if( K/(float) nbN[whichPop(i)] >= randkernel(state, kNeuron)) // neuron[id] receives input from j ?
	dev_conVec[id + i * maxNeurons] = 1; 
}

__global__ void KernelGenDistDepConMat(hiprandState *state, float *dev_conVec, int lChunck, int maxNeurons) {
  /* GENERATE CONNECTION MATRIX WITH ANOTOMIC CONNECTIVITY PROFILE */
  /* indexing of matrix row + clm x N_NEURONS*/
  unsigned long int id =  (unsigned long int)threadIdx.x + blockIdx.x * blockDim.x;
  unsigned long int kNeuron = id + lChunck * maxNeurons, i;

  if(id < maxNeurons & kNeuron < N_NEURONS)
    for(i=0; i<N_NEURONS; i++) 
      if(dev_conVec[id + i * maxNeurons] >= randkernel(state, kNeuron)) /* neuron[id] receives input from i ? */
	dev_conVec[id + i * maxNeurons] = 1 ;
      else
	dev_conVec[id + i * maxNeurons] = 0 ; 
}


int main(int argc, char *argv[]) {

  int N = N_NEURONS ;

  int *nbN, *Cpt ;
  nbNeurons(N, nbN);
  CptNeurons(nbN, Cpt);
  
  // ///////////////////////////////////////////////////////////////////    
  
  int nChunks = 1, deviceId = 0, maxNeurons = N_NEURONS ;

  ///////////////////////////////////////////////////////////////////

  hipDeviceProp_t prop;
  unsigned long maxMem = 12079136768;

  cudaCheck(hipGetDeviceProperties(&prop, deviceId));
  printf("Global Mem = %ld, ", prop.totalGlobalMem);
  maxMem = prop.totalGlobalMem;

  if(maxMem < (N_NEURONS * N_NEURONS * 4 + N_NEURONS * 4)) {
    while( maxMem < ( (N_NEURONS / nChunks) * N_NEURONS * 4   + N_NEURONS * 5 ) ) 
      nChunks += 1 ;
    
    if( nChunks % 2 !=0 )
      nChunks += 1 ;
  }
  
  maxNeurons = N_NEURONS / nChunks;

  if(IF_CHUNKS) {
    nChunks = NCHUNKS ;
    maxNeurons = MAXNEURONS ;
  }

  printf(" maxNeurons = %d, nChunks = %d\n", maxNeurons, nChunks);

  ///////////////////////////////////////////////////////////////////

  /* choose 256 threads per block for high occupancy */
  int ThreadsPerBlock = 512 ;
  int BlocksPerGrid = ( N_NEURONS + ThreadsPerBlock-1 ) / ThreadsPerBlock;
  
  if(BlocksPerGrid > 65536) {
    printf("BlocksPerGrid exceds valid number of allowed blocks of 65536");
    exit(-1);
  }

  ///////////////////////////////////////////////////////////////////

  unsigned long long int chunckSize = ( (unsigned long long) N_NEURONS / nChunks) * N_NEURONS ;
  printf("chunckSize = %llu, ", chunckSize);
  BlocksPerGrid = (maxNeurons + ThreadsPerBlock - 1) / ThreadsPerBlock;
  printf("Threads per block : %d, Blocks per grid : %d \n", ThreadsPerBlock, BlocksPerGrid);

  // ///////////////////////////////////////////////////////////////////    

  float *dev_conVecPtr, *dev_preFactor ; //*preFactor = NULL;
  float *fullConVec = NULL, *conVec = NULL ;

  // int *dev_IdPost, *dev_nbPost , *dev_nbPreSab ;
  // int *host_IdPost, *host_nbPost, *host_nbPreSab ;
  int *IdPost, *nbPost ;

  ///////////////////////////////////////////////////////////////////

  hiprandState *devStates;
  
  fullConVec = (float *) malloc((unsigned long long) N_NEURONS * N_NEURONS * sizeof(float));

  IdPost = (int *) malloc((unsigned long long)N_NEURONS * (2ULL + (unsigned long long)K + N_NEURONS) * sizeof(int));
  nbPost = (int *) malloc((unsigned long long) N_NEURONS * sizeof(int));

  int **nbPreSab = (int **)malloc(nbpop * sizeof(int *));
  for(int i=0; i<nbpop; i++)
    nbPreSab[i] = (int *) malloc(nbpop * sizeof(int));

  for(int i=0; i<nbpop; i++)
    for(int j=0;j<nbpop;j++)
      nbPreSab[i][j] = 0 ;
    
  ////////////////////////////////////////////////////////////////////    

  cudaCheck(hipMalloc((void **)&devStates,  N_NEURONS * sizeof(hiprandState)));

  cudaCheck(hipHostMalloc((void **)&conVec, (N_NEURONS / nChunks) * N_NEURONS * sizeof(float)));

  cudaCheck(hipMalloc((void **)&dev_conVecPtr, (N_NEURONS / nChunks) * N_NEURONS * sizeof(float)));

  cudaCheck(hipMalloc((void **)&dev_preFactor, 2 * N_NEURONS * sizeof(float)));

  // cudaCheck(hipHostMalloc((void **)&preFactor, 2 * N_NEURONS * sizeof(float)));

  // cudaCheck(hipMalloc((void **)&dev_IdPost,  N_NEURONS * sizeof(int)));

  // cudaCheck(hipMalloc((void **)&dev_nbPost, N_NEURONS * sizeof(int)));

  // cudaCheck(hipMalloc((void **)&dev_nbPreSab, nbpop * nbpop * sizeof(int)));

  // cudaCheck(hipHostMalloc((void **)&host_IdPost,  N_NEURONS * sizeof(int)));

  // cudaCheck(hipHostMalloc((void **)&host_nbPost, N_NEURONS * sizeof(int)));

  // cudaCheck(hipHostMalloc((void **)&host_nbPreSab, nbpop * nbpop * sizeof(int)));

  ///////////////////////////////////////////////////////////////////

  enum ConMat_type {
    random,distDependent,bump
  };

  ConMat_type conMatType = random ; 
  if(IF_SPACE) {
    printf("Generating Spatial Matrix ... \n") ; 
    conMatType = distDependent ;
  }
  else 
    if(IF_BUMP)
      printf("Generating Random Matrix with specific connections ... \n") ; 
    else
      printf("Generating Random Matrix ... \n") ; 
      

  printf("Setup kernel ... \n");
  setup_kernel<<<BlocksPerGrid, ThreadsPerBlock>>>(devStates, time(NULL));
  cudaCheckLastError("setup_kernel failed\n");

  ///////////////////////////////////////////////////////////////////
  
  double *host_Sigma ;
  cudaCheck(hipHostMalloc((void **)&host_Sigma,  nbpop * sizeof(double))); 
  for(int j=0;j<nbpop;j++) 
    host_Sigma[j] = Sigma[j] ;
  
  if(IF_SPACE || IF_BUMP) {
    printf("Sigma ") ;
    for(int j=0;j<nbpop;j++) 
      printf("%.4f ",Sigma[j]) ;
    printf("\n") ;
  }

  int counter = 0 ;

  switch(conMatType) {
    
  case random:
    
    for(unsigned long long int i = 0; i < nChunks; i++) { 
      
      printf("Generating chunk %llu ... \n", i) ; fflush(stdout) ;
      
      initConVec<<<BlocksPerGrid, ThreadsPerBlock>>>(dev_conVecPtr, maxNeurons);

      // initIdPost<<<BlocksPerGrid, ThreadsPerBlock>>>(dev_IdPost);
      // initNbPost<<<BlocksPerGrid, ThreadsPerBlock>>>(dev_nbPost);
      // initNbPreS<<<BlocksPerGrid, ThreadsPerBlock>>>(dev_nbPreSab);

      printf(" Generating Binary Matrix ...\n") ;
      if(IF_BUMP) {
	KernelGenConRing<<<BlocksPerGrid, ThreadsPerBlock>>>(dev_conVecPtr,i,maxNeurons,nbN,Cpt,host_Sigma); 
	KernelGenDistDepConMat<<<BlocksPerGrid, ThreadsPerBlock>>>(devStates, dev_conVecPtr, i, maxNeurons) ; 
      }
      else
	kernelGenConMat<<<BlocksPerGrid, ThreadsPerBlock>>>(devStates, dev_conVecPtr, i, maxNeurons, nbN); 
      
      printf("  Copy dev to Host ... \n") ;
      cudaCheck(hipMemcpy(conVec, dev_conVecPtr, ( N_NEURONS/ nChunks ) * N_NEURONS * sizeof(float), hipMemcpyDeviceToHost)) ;
      
      for(unsigned long long int j = 0; j < chunckSize ; j++) 
	fullConVec[j + chunckSize * i] = conVec[j] ; 

      // printf("   Generating Sparse Vectors ...\n") ;
      // GenSparseRep<<<BlocksPerGrid, ThreadsPerBlock>>>(dev_conVecPtr, dev_IdPost, dev_nbPost, dev_nbPreSab, i, maxNeurons);

      // printf("    Copy dev to Host ... \n") ;
      // cudaCheck(hipMemcpy(host_IdPost, dev_IdPost, N_NEURONS * sizeof(int), hipMemcpyDeviceToHost)) ;
      // cudaCheck(hipMemcpy(host_nbPost, dev_nbPost, N_NEURONS * sizeof(int), hipMemcpyDeviceToHost)) ;
      // cudaCheck(hipMemcpy(host_nbPreSab, dev_nbPreSab, nbpop * nbpop * sizeof(int), hipMemcpyDeviceToHost)) ;


      // // for(unsigned long long int j = 0; j < N_NEURONS ; j++) 
      // // 	if( host_IdPost[j] !=0 ) {
      // // 	  IdPost[counter] = host_IdPost[j] ; 
      // // 	  counter+=1 ;
      // // 	}
      
      // for(unsigned long long int j = 0; j < N_NEURONS ; j++) 
      // 	IdPost[j + chunckSize * i] = host_IdPost[j] ; 
      
      // for(unsigned long int j = 0; j < N_NEURONS ;j++) {
      // 	nbPost[j] += host_nbPost[j] ; 
      // 	nbPreSab[whichPop(j)][0] += host_nbPost[j] ;
      // }

      // for(int j=0;j<nbpop;j++)
      // 	for(int k=0;k<nbpop;k++)
      // 	  nbPreSab[j][k] += host_nbPreSab[j + nbpop * k] ;
    }
    
    break;
    
  case distDependent:
      
    initPreFactor<<<BlocksPerGrid, ThreadsPerBlock>>>(dev_preFactor);
    
    for(unsigned long long int i = 0; i < nChunks; i++) { 

      initConVec<<<BlocksPerGrid, ThreadsPerBlock>>>(dev_conVecPtr, maxNeurons);

      // initIdPost<<<BlocksPerGrid, ThreadsPerBlock>>>(dev_IdPost);
      // initNbPost<<<BlocksPerGrid, ThreadsPerBlock>>>(dev_nbPost);
      // initNbPreS<<<BlocksPerGrid, ThreadsPerBlock>>>(dev_nbPreSab);

      printf("Generating chunk %llu ... \n", i); fflush(stdout);
	
      printf(" Generating Probabilty Matrix ...\n");
      KernelGenConProbMat<<<BlocksPerGrid, ThreadsPerBlock>>>(dev_conVecPtr,i,maxNeurons,nbN,Cpt,host_Sigma); 
      
      printf("  Generating preFactor ...\n");
      KernelConProbPreFactor<<<BlocksPerGrid, ThreadsPerBlock>>>(dev_conVecPtr, dev_preFactor, i, maxNeurons) ;      

      printf("   Copy dev to Host ...\n") ;
      cudaCheck(hipMemcpy(conVec, dev_conVecPtr, ( N_NEURONS/ nChunks ) * N_NEURONS * sizeof(float), hipMemcpyDeviceToHost)) ;

      for(unsigned long long int j = 0; j < chunckSize ; j++) {
	fullConVec[j + chunckSize * i] = conVec[j] ; 
	
	// if(conVec[j]!=1) {
	//   printf("\n ERRROR Chunk %llu conVec[%llu] = %.3f \n", i, j, conVec[j] ) ;
	//   exit(-1) ;
	// }
	
	conVec[j] = 0 ;
      }      

    }
    
    // printf("Copy preFactor to Host ...") ; 
    // cudaCheck(hipMemcpy(preFactor, dev_preFactor, 2 * N_NEURONS * sizeof(float), hipMemcpyDeviceToHost) ) ; 
    // printf(" Done\n ") ; 
    
    // printf(" Check preFactor ...") ; 
    // for(int j=0;j<2*N_NEURONS;j++)
    //   if(preFactor[j]!=N_NEURONS/nbpop) {
    // 	printf("ERROR clm %d prefactor %.3f \n", j, preFactor[j]) ;
    // 	exit(-1) ;
    //   }
    // printf(" %.0f ", preFactor[0]) ;
    // printf(" Done\n") ; 

    for(unsigned long long int i = 0; i < nChunks; i++) { 

      printf("Generating chunk %llu ... \n", i); fflush(stdout);
      
      initConVec<<<BlocksPerGrid, ThreadsPerBlock>>>(dev_conVecPtr, maxNeurons);

      for(unsigned long long int j = 0; j < chunckSize ; j++) 
	conVec[j] = fullConVec[j + chunckSize * i] ; 

      printf("\n Copy Host to dev ...\n") ;
      cudaCheck(hipMemcpy(dev_conVecPtr, conVec, ( N_NEURONS/ nChunks ) * N_NEURONS * sizeof(float), hipMemcpyHostToDevice)) ;
      
      printf("  Generating Normalized Matrix ...\n") ;
      KernelConProbNorm<<<BlocksPerGrid, ThreadsPerBlock>>>(dev_conVecPtr, dev_preFactor, i, maxNeurons) ;
            
      printf("   Generating Binary Matrix ...\n") ;
      KernelGenDistDepConMat<<<BlocksPerGrid, ThreadsPerBlock>>>(devStates, dev_conVecPtr, i, maxNeurons) ; 
            
      // printf("    Generating Sparse Vectors ...\n") ;
      // GenSparseRep<<<BlocksPerGrid, ThreadsPerBlock>>>(dev_conVecPtr, dev_IdPost, dev_nbPost, dev_nbPreSab, i, maxNeurons);

      // printf("  Copy dev to Host ...\n") ;
      // cudaCheck(hipMemcpy(host_IdPost, dev_IdPost, N_NEURONS * sizeof(int), hipMemcpyDeviceToHost)) ;
      // cudaCheck(hipMemcpy(host_nbPost, dev_nbPost, N_NEURONS * sizeof(int), hipMemcpyDeviceToHost)) ;
      // cudaCheck(hipMemcpy(host_nbPreSab, dev_nbPreSab, nbpop * nbpop * sizeof(int), hipMemcpyDeviceToHost)) ;

      // for(unsigned long long int j = 0; j < N_NEURONS ; j++) 
      // 	if(host_IdPost[j] !=0 ) {
      // 	  IdPost[counter] = host_IdPost[j] ; 
      // 	  counter+=1 ;
      // 	}
      
      // for(unsigned long int j = 0; j < N_NEURONS ;j++) 
      // 	nbPost[j] += host_nbPost[j] ; 

      // for(int j=0;j<nbpop;j++)
      // 	for(int k=0;k<nbpop;k++)
      // 	  nbPreSab[j][k] += host_nbPreSab[j + nbpop * k] ;

      cudaCheck(hipMemcpy(conVec, dev_conVecPtr, ( N_NEURONS/ nChunks ) * N_NEURONS * sizeof(float), hipMemcpyDeviceToHost)) ;
      
      for(unsigned long long int j = 0; j < chunckSize ; j++) {

	// if(normConVec[j]!=N_NEURONS/nbpop) {
	//   printf("\n ERRROR Chunk %llu normConVec[%llu] = %.0f \n", i, j, conVec[j] ) ;
	//   exit(-1) ;
	// }

	fullConVec[j + chunckSize * i] = conVec[j] ; 	
	conVec[j] = 0 ;
      }

    }

    break;    

  case bump:
    break ;
    
  default:
    for(unsigned long long int i = 0; i < nChunks; i++) 
      kernelGenConMat<<<BlocksPerGrid, ThreadsPerBlock>>>(devStates, dev_conVecPtr, i, maxNeurons, nbN);
  }
  
  printf("Free devPtr ... ");

  hipFree(dev_conVecPtr); 
  hipFree(dev_preFactor); 

  hipHostFree(host_Sigma); 
  hipHostFree(conVec); 

  // hipFree(dev_IdPost);  
  // hipFree(dev_nbPost);
  // hipFree(dev_nbPreSab);

  // hipHostFree(host_IdPost);  
  // hipHostFree(host_nbPost);
  // hipHostFree(host_nbPreSab);
  // // hipHostFree(preFactor); 

  printf("Done\n") ;

  ///////////////////////////////////////////////////////////////////    

  // ///////////////////////////////////////////////////////////////////    
  // // On CPU 
  // ///////////////////////////////////////////////////////////////////    

  // ///////////////////////////////////////////////////////////////////    
    
  unsigned long int *idxPost = (unsigned long int *) malloc( N * sizeof(unsigned long int) ); // idx of the post neurons 
  idxPost[0] = 0 ;
  
  ///////////////////////////////////////////////////////////////////    
  // Average number of Presynaptic neurons
  ///////////////////////////////////////////////////////////////////    

  char *path = '\0';
  CreatePath(path,N) ;
  
  CheckPres(path,nbN,nbPreSab) ;
  free(nbPreSab);

  ///////////////////////////////////////////////////////////////////    
  // Writing to File
  ///////////////////////////////////////////////////////////////////

  WritetoFile(path,N,IdPost,nbPost,idxPost) ;
  CheckSparseVec(path,IdPost,nbPost,idxPost) ;

  free(IdPost);
  free(idxPost);
  free(nbPost);

  ///////////////////////////////////////////////////////////////////    
  // Writing Complete Matrix
  ///////////////////////////////////////////////////////////////////

  if(IF_MATRIX)
    WriteMatrix(path,fullConVec) ;

  printf("Free Host ptr ... ") ;

  hipHostFree(nbN); 
  hipHostFree(Cpt); 

  free(fullConVec);

  printf("Done\n") ;

  return 0 ;
  
}