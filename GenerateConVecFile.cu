#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "mycurand.h"
#include "librairies.h"
#include "cuPrintf.cu"
#include "devFunctionProtos.h"
#include "devHostConstants.h"
#include "Matrix_Utils.cu"
#include "GenConProbDistDepMat.cu"

///////////////////////////////////////////////////////////////////    

void __cudaCheck(hipError_t err, const char* file, const int line);
#define cudaCheck(err) __cudaCheck (err, __FILE__, __LINE__)

void __cudaCheckLastError(const char* errorMessage, const char* file, const int line);
#define cudaCheckLastError(msg) __cudaCheckLastError (msg, __FILE__, __LINE__)

void __cudaCheck(hipError_t err, const char *file, const int line) {
  if( hipSuccess != err) {
    fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
      file, line, (int)err, hipGetErrorString( err ) );
    exit(-1);
  }
}

void __cudaCheckLastError(const char *errorMessage, const char *file, const int line) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
      file, line, errorMessage, (int)err, hipGetErrorString( err ) );
    exit(-1);
  }
}

///////////////////////////////////////////////////////////////////    

__host__ void nbNeurons(int* &nbN) {
  
  cudaCheck(hipHostMalloc((void **)&nbN, nbpop * sizeof(int)));
  printf("Number of neurons : ") ;
  int i = 0;
  while(i<nbpop) {
    if(i==0)
      nbN[i] = N_NEURONS*popSize ;
    else
      nbN[i] = ( N_NEURONS * (100 - int( popSize * 100 ) ) / 100 ) / max( (nbpop-1), 1 ) ;
    
    printf("%d ", nbN[i]) ;
    ++i ;
  }
  printf("\n") ;
}

///////////////////////////////////////////////////////////////////    
 
__host__ void CptNeurons(int* nbN, int* &Cpt) {
  cudaCheck(hipHostMalloc((void **)&Cpt, nbpop * sizeof(int)));
  printf("Counter : ") ;
  for(int i=0;i<nbpop+1;i++) {
    Cpt[i] = 0 ;
    for(int j=0;j<i;j++) {
      Cpt[i] = Cpt[i] + nbN[j] ; 
    }
    printf("%d ", Cpt[i]) ;
  }
  printf("\n") ;
}

///////////////////////////////////////////////////////////////////    

__global__ void initConVec(float *dev_conVec, int maxNeurons) {
  unsigned long int id = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned long int i;
  if(id < maxNeurons) 
    for(i = 0; i < N_NEURONS; i++) 
      dev_conVec[i + id * N_NEURONS] = 0 ; 
}

///////////////////////////////////////////////////////////////////    

__global__ void initPreFactor(float *dev_preFactor) {
  unsigned long int i;
  for(i = 0; i < 2 * N_NEURONS ; i++) 
    dev_preFactor[i] = 0 ; 
}

__global__ void setup_kernel(hiprandState *state, unsigned long long seed ) {
  unsigned long int id = threadIdx.x + blockIdx.x * blockDim.x;
  /* Each thread gets different seed, a different sequence number, no offset */
  if(id < N_NEURONS) 
    hiprand_init(seed * (id + 7), id, 0, &state[id]);
}

__device__ float randkernel(hiprandState *state, unsigned long int kNeuron) {
  /*RETURNS ONE SAMPLE FROM UNIFORM DISTRIBUTION*/
  /*  unsigned int id = (unsigned long int)threadIdx.x + blockIdx.x * blockDim.x;*/
  float randNumber= 0.0;
  if(kNeuron < N_NEURONS) {
    hiprandState localState = state[kNeuron]; /* state in global memory */
    randNumber = hiprand_uniform(&localState);
    state[kNeuron] = localState;
  }
  return randNumber;
}

///////////////////////////////////////////////////////////////////    

__global__ void kernelGenConMat(hiprandState *state, float *dev_conVec, int lChunck, int maxNeurons, int* nbN) {

  /* indexing of matrix row + clm x N_NEURONS*/
  unsigned long id =  (unsigned long int)threadIdx.x + blockIdx.x * blockDim.x;
  unsigned long int kNeuron = id + lChunck * maxNeurons;
  unsigned long int i;
  
  if(id < maxNeurons && kNeuron < N_NEURONS) 
    for(i=0; i<N_NEURONS; i++) {// j is row and id is clmn 
      // cuPrintf("id %d i %d \n",id,i) ;
      if( K/(float) nbN[whichPop(kNeuron)] >= randkernel(state, kNeuron)) // neuron[id] receives input from j
	dev_conVec[id + i * maxNeurons] = 1 ;
      else
	dev_conVec[id + i * maxNeurons] = 0 ;
    }
}

///////////////////////////////////////////////////////////////////    

__global__ void KernelGenConRing(hiprandState *state, float *dev_conVec, int lChunck, int maxNeurons, int *nbN, int *Cpt, const double *Sigma) {

  unsigned long id =  (unsigned long int)threadIdx.x + blockIdx.x * blockDim.x;
  unsigned long int kNeuron = id + lChunck * maxNeurons ;
  unsigned long int i;
  double xa, xb;
  
  if(id < maxNeurons && kNeuron < N_NEURONS) { 
    xa = XCordinate(kNeuron,nbN,Cpt) ; // Mij column to row 
    for(i=0; i < N_NEURONS; i++) { // i-->id column to row, P[row][clm] = G(X[row],X[clm],Sigma[clm]) 
      xb = XCordinate(i,nbN,Cpt) ;
      dev_conVec[id + i * maxNeurons] = (float) ( K / (float) nbN[whichPop(i)] ) * ( 1.0 + 2.0 * Sigma[whichPop(i)] * Sigma[whichPop(kNeuron)] / sqrt(K) * cos( 2.0 * M_PI * (xa-xb) ) ) ;
      
      if( dev_conVec[id + i * maxNeurons] >= randkernel(state,kNeuron)) // neuron[id] receives input from j ?
	dev_conVec[id + i * maxNeurons] = 1; 
      else
	dev_conVec[id + i * maxNeurons] = 0; 	
      
    }
  }
}

///////////////////////////////////////////////////////////////////    

__global__ void KernelGenDistDepConMat(hiprandState *state, float *dev_conVec, int lChunck, int maxNeurons) {
  /* GENERATE CONNECTION MATRIX WITH ANOTOMIC CONNECTIVITY PROFILE */
  /* indexing of matrix row + clm x N_NEURONS*/
  unsigned long int id =  (unsigned long int)threadIdx.x + blockIdx.x * blockDim.x;
  unsigned long int kNeuron = id + lChunck * maxNeurons, i;

  if(id < maxNeurons && kNeuron < N_NEURONS)
    for(i=0; i<N_NEURONS; i++) 

      if(IF_SPEC) 
	
	if(true)

	  if( ( K -sqrt(K) ) / (float) ( N_NEURONS/nbpop ) + dev_conVec[id + i * maxNeurons] >= randkernel(state, kNeuron)) /* neuron[id] receives input from i ? */
	    dev_conVec[id + i * maxNeurons] = 1. ;
	  else
	    dev_conVec[id + i * maxNeurons] = 0. ; 

	else
	  if( K  / (float) ( N_NEURONS/nbpop ) >= randkernel(state, kNeuron)) /* neuron[id] receives input from i ? */
	    dev_conVec[id + i * maxNeurons] = 1. ;
	  else
	    dev_conVec[id + i * maxNeurons] = 0. ;  
  
      else

	if(dev_conVec[id + i * maxNeurons] >= randkernel(state, kNeuron)) /* neuron[id] receives input from i ? */
	  dev_conVec[id + i * maxNeurons] = 1. ;
	else
	  dev_conVec[id + i * maxNeurons] = 0. ; 
}


///////////////////////////////////////////////////////////////////    

int main(int argc, char *argv[]) {

  int N = N_NEURONS ;
  
  int *nbN, *Cpt ;
  nbNeurons(nbN) ;
  CptNeurons(nbN, Cpt) ;
  
  // ///////////////////////////////////////////////////////////////////    
  
  int nChunks = 1, deviceId = 0, maxNeurons = N_NEURONS ;

  ///////////////////////////////////////////////////////////////////

  hipDeviceProp_t prop;
  unsigned long maxMem = 12079136768;

  cudaCheck(hipGetDeviceProperties(&prop, deviceId));
  printf("Global Mem = %ld, ", prop.totalGlobalMem);
  maxMem = prop.totalGlobalMem;

  if(maxMem < (N_NEURONS * N_NEURONS * 4 + N_NEURONS * 4)) {
    while( maxMem < ( (N_NEURONS / nChunks) * N_NEURONS * 4   + N_NEURONS * 5 ) ) 
      nChunks += 1 ;
    
    if( nChunks % 2 !=0 )
      nChunks += 1 ;
  }
  
  maxNeurons = N_NEURONS / nChunks;

  if(IF_CHUNKS) {
    nChunks = NCHUNKS ;
    maxNeurons = MAXNEURONS ;
  }

  printf(" maxNeurons = %d, nChunks = %d\n", maxNeurons, nChunks);

  ///////////////////////////////////////////////////////////////////

  /* choose 256 threads per block for high occupancy */
  int ThreadsPerBlock = N_THREADS ;
  int BlocksPerGrid = ( N_NEURONS + ThreadsPerBlock-1 ) / ThreadsPerBlock;
  
  if(BlocksPerGrid > 65536) {
    printf("BlocksPerGrid exceds valid number of allowed blocks of 65536");
    exit(-1);
  }

  hiprandState *devStates;
  cudaCheck(hipMalloc((void **)&devStates,  N_NEURONS * sizeof(hiprandState)));
  
  printf("Setup kernel ... \n");
  setup_kernel<<<BlocksPerGrid, ThreadsPerBlock>>>(devStates, time(NULL));
  cudaCheckLastError("setup_kernel failed\n");

  ///////////////////////////////////////////////////////////////////

  unsigned long long int chunckSize = ( (unsigned long long) N_NEURONS / nChunks) * N_NEURONS ;
  printf("chunckSize = %llu, ", chunckSize);
  BlocksPerGrid = (maxNeurons + ThreadsPerBlock - 1) / ThreadsPerBlock;
  printf("Threads per block : %d, Blocks per grid : %d \n", ThreadsPerBlock, BlocksPerGrid);

  // ///////////////////////////////////////////////////////////////////    

  float *dev_conVecPtr, *dev_preFactor ; //*preFactor = NULL;
  float *fullConVec = NULL, *conVec = NULL ;
  int *IdPost, *nbPost ;

  ///////////////////////////////////////////////////////////////////
  
  fullConVec = (float *) malloc((unsigned long long) N_NEURONS * N_NEURONS * sizeof(float));

  IdPost = (int *) malloc((unsigned long long) N_NEURONS * (2ULL + (unsigned long long)K + N_NEURONS) * sizeof(int));
  nbPost = (int *) malloc((unsigned long long) N_NEURONS * sizeof(int));

  int **nbPreSab = (int **)malloc(nbpop * sizeof(int *) );
  for(int i=0; i<nbpop; i++)
    nbPreSab[i] = (int *) malloc(nbpop * sizeof(int) );

  for(int i=0; i<nbpop; i++)
    for(int j=0;j<nbpop;j++)
      nbPreSab[i][j] = 0 ;
  
  ////////////////////////////////////////////////////////////////////    

  cudaCheck(hipHostMalloc((void **)&conVec, (N_NEURONS / nChunks) * N_NEURONS * sizeof(float)));

  cudaCheck(hipMalloc((void **)&dev_conVecPtr, (N_NEURONS / nChunks) * N_NEURONS * sizeof(float)));

  cudaCheck(hipMalloc((void **)&dev_preFactor, 2 * N_NEURONS * sizeof(float)));

  // cudaCheck(hipHostMalloc((void **)&preFactor, 2 * N_NEURONS * sizeof(float)));

  ///////////////////////////////////////////////////////////////////

  enum ConMat_type {
    random,distDependent
  };

  ConMat_type conMatType = random ; 
  if(IF_SPACE) {
    printf("Generating Spatial Matrix ... \n") ; 
    conMatType = distDependent ;
  }
  else 
    if(IF_RING) 
      printf("Generating Ring ... \n") ; 
    else 
      printf("Generating Random Matrix ... \n") ; 
  
  if(IF_SPEC) 
    printf("with specific connections ... \n") ; 
  
  ///////////////////////////////////////////////////////////////////
  
  double *host_Sigma ;
  cudaCheck(hipHostMalloc((void **)&host_Sigma,  nbpop * sizeof(double))); 
  for(int j=0;j<nbpop;j++) 
    host_Sigma[j] = Sigma[j] ;
  
  if(IF_SPACE || IF_RING) {
    printf("Sigma ") ;
    for(int j=0;j<nbpop;j++) 
      printf("%.4f ",Sigma[j]) ;
    printf("\n") ;
  }

  cudaPrintfInit();

  switch(conMatType) {
    
  case random:
    
    for(unsigned long long int i = 0; i < nChunks; i++) { 

      initConVec<<<BlocksPerGrid, ThreadsPerBlock>>>(dev_conVecPtr, maxNeurons);
      
      printf("Generating chunk %llu ... \n", i) ; fflush(stdout) ;
      
      printf(" Generating Binary Matrix ...\n") ;
      if(IF_RING) 
	KernelGenConRing<<<BlocksPerGrid, ThreadsPerBlock>>>(devStates,dev_conVecPtr,i,maxNeurons,nbN,Cpt,host_Sigma) ; 
      else {
	kernelGenConMat<<<BlocksPerGrid, ThreadsPerBlock>>>(devStates, dev_conVecPtr, i, maxNeurons, nbN); 
	cudaPrintfDisplay(stdout, true);
      }
      
      printf("  Copy dev to Host ... \n") ;
      cudaCheck(hipMemcpy(conVec, dev_conVecPtr, ( N_NEURONS / nChunks ) * N_NEURONS * sizeof(float), hipMemcpyDeviceToHost)) ;
      
      for(unsigned long long int j = 0; j < chunckSize ; j++) {
	fullConVec[j + chunckSize * i] = (float) conVec[j] ; 
	// printf("# %llu Con %f fullConVec %f \n", j + chunckSize * i, conVec[j], fullConVec[j + chunckSize * i]) ;
	conVec[j] = 0 ;
      }
    }

    cudaPrintfEnd();

    break;
    
  case distDependent:
      
    initPreFactor<<<BlocksPerGrid, ThreadsPerBlock>>>(dev_preFactor);
    
    for(unsigned long long int i = 0; i < nChunks; i++) { 

      initConVec<<<BlocksPerGrid, ThreadsPerBlock>>>(dev_conVecPtr, maxNeurons);

      printf("Generating chunk %llu ... \n", i); fflush(stdout);
	
      printf(" Generating Probabilty Matrix ...\n");
      KernelGenConProbMat<<<BlocksPerGrid, ThreadsPerBlock>>>(dev_conVecPtr,i,maxNeurons,nbN,Cpt,host_Sigma); 
      
      printf("  Generating preFactor ...\n");
      KernelConProbPreFactor<<<BlocksPerGrid, ThreadsPerBlock>>>(dev_conVecPtr, dev_preFactor, i, maxNeurons) ;      

      printf("   Copy dev to Host ...\n") ;
      cudaCheck(hipMemcpy(conVec, dev_conVecPtr, ( N_NEURONS/ nChunks ) * N_NEURONS * sizeof(float), hipMemcpyDeviceToHost)) ;

      for(unsigned long long int j = 0; j < chunckSize ; j++) {
	fullConVec[j + chunckSize * i] = conVec[j] ; 
	
	// if(conVec[j]!=1) {
	//   printf("\n ERRROR Chunk %llu conVec[%llu] = %.3f \n", i, j, conVec[j] ) ;
	//   exit(-1) ;
	// }
	
	conVec[j] = 0 ;
      }      

    }
    
    // printf("Copy preFactor to Host ...") ; 
    // cudaCheck(hipMemcpy(preFactor, dev_preFactor, 2 * N_NEURONS * sizeof(float), hipMemcpyDeviceToHost) ) ; 
    // printf(" Done\n ") ; 
    
    // printf(" Check preFactor ...") ; 
    // for(int j=0;j<2*N_NEURONS;j++)
    //   if(preFactor[j]!=N_NEURONS/nbpop) {
    // 	printf("ERROR clm %d prefactor %.3f \n", j, preFactor[j]) ;
    // 	exit(-1) ;
    //   }
    // printf(" %.0f ", preFactor[0]) ;
    // printf(" Done\n") ; 

    for(unsigned long long int i = 0; i < nChunks; i++) { 

      printf("Generating chunk %llu ... \n", i); fflush(stdout);
      
      initConVec<<<BlocksPerGrid, ThreadsPerBlock>>>(dev_conVecPtr, maxNeurons);

      for(unsigned long long int j = 0; j < chunckSize ; j++) 
	conVec[j] = fullConVec[j + chunckSize * i] ; 

      printf("\n Copy Host to dev ...\n") ;
      cudaCheck(hipMemcpy(dev_conVecPtr, conVec, ( N_NEURONS/ nChunks ) * N_NEURONS * sizeof(float), hipMemcpyHostToDevice)) ;
      
      printf("  Generating Normalized Matrix ...\n") ;
      KernelConProbNorm<<<BlocksPerGrid, ThreadsPerBlock>>>(dev_conVecPtr, dev_preFactor, i, maxNeurons) ;
            
      printf("   Generating Binary Matrix ...\n") ;
      KernelGenDistDepConMat<<<BlocksPerGrid, ThreadsPerBlock>>>(devStates, dev_conVecPtr, i, maxNeurons) ; 
            
      cudaCheck(hipMemcpy(conVec, dev_conVecPtr, ( N_NEURONS/ nChunks ) * N_NEURONS * sizeof(float), hipMemcpyDeviceToHost)) ;
      
      for(unsigned long long int j = 0; j < chunckSize ; j++) {

	// if(normConVec[j]!=N_NEURONS/nbpop) {
	//   printf("\n ERRROR Chunk %llu normConVec[%llu] = %.0f \n", i, j, conVec[j] ) ;
	//   exit(-1) ;
	// }

	fullConVec[j + chunckSize * i] = conVec[j] ; 	
	conVec[j] = 0 ;
      }

    }
    
    break;    
    
  default:
    for(unsigned long long int i = 0; i < nChunks; i++) 
      kernelGenConMat<<<BlocksPerGrid, ThreadsPerBlock>>>(devStates, dev_conVecPtr, i, maxNeurons, nbN);
  }
  
  printf("Free devPtr ... ");

  hipFree(dev_conVecPtr); 
  hipFree(dev_preFactor); 

  hipHostFree(host_Sigma); 
  hipHostFree(conVec); 

  printf("Done\n") ;

  ///////////////////////////////////////////////////////////////////    

  // ///////////////////////////////////////////////////////////////////    
  // // On CPU 
  // ///////////////////////////////////////////////////////////////////    

  // ///////////////////////////////////////////////////////////////////    
    
  unsigned long int *idxPost = (unsigned long int *) malloc( N * sizeof(unsigned long int) ); // idx of the post neurons 
  idxPost[0] = 0 ;

  printf("Generating vectors nbPost & IdPost ... ");

  int counter = 0 ;
  
  for(int i=0;i<nbpop;i++) 
    for(int k=Cpt[i];k<Cpt[i+1];k++) { //Presynaptic neurons
      for(int j=0;j<nbpop;j++) 
  	for(int l=Cpt[j];l<Cpt[j+1];l++) //Postsynaptic neurons
  	  if(fullConVec[k + N_NEURONS * l]) { // k-->l column to row
	    IdPost[counter] = l ;
	    nbPost[k]++ ;
	    nbPreSab[j][i]++ ;
  	    counter+=1 ;
  	  }   
      // printf("PresId %d, nPost %d \r",k,nbPost[k]);
    }
  
  ///////////////////////////////////////////////////////////////////    
  // Average number of Presynaptic neurons
  ///////////////////////////////////////////////////////////////////    

  char *path = '\0';
  CreatePath(path,N) ;
  
  CheckPres(path,nbN,nbPreSab) ;
  free(nbPreSab);

  ///////////////////////////////////////////////////////////////////    
  // Writing to File
  ///////////////////////////////////////////////////////////////////

  WritetoFile(path,N,IdPost,nbPost,idxPost) ;

  free(IdPost);
  free(idxPost);
  free(nbPost);

  // CheckSparseVec(path) ;
 
  ///////////////////////////////////////////////////////////////////    
  // Writing Complete Matrix
  ///////////////////////////////////////////////////////////////////

  if(IF_MATRIX)
    WriteMatrix(path,fullConVec) ;

  printf("Free Host ptr ... ") ;

  hipHostFree(nbN); 
  hipHostFree(Cpt); 

  free(fullConVec);

  printf("Done\n") ;

  return 0 ;
  
}