#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include "librairies.h"
#include "CudaFunc.cu"
#include "cuPrintf.cu"

#include "ImportSpikes.cu"
#include "CrossCorr.cu"

#define Nx 2620UL
#define Ny 2620UL
#define BATCH 1

#define N_THREADS 256

///////////////////////////////////////////////

__host__ __device__ hipfftComplex ComplexMul(hipfftComplex a, hipfftComplex b) {
  
  hipfftComplex c;
  c.x = a.x * b.x - a.y * b.y;  
  c.y = a.x * b.y + a.y * b.x;  

  return c ;
}

///////////////////////////////////////////////

__host__ float Norm(float* x, unsigned long Size) {
  unsigned long i ;
  float y=0 ;
  for(i=0;i<Size;i++)
    y += x[i]*x[i] ;

  return sqrt(y) ;

}
///////////////////////////////////////////////

__global__ void cudaMultiply(hipfftComplex *dev_xdata, hipfftComplex *dev_ydata, hipfftComplex *dev_zdata, unsigned long dataSize) {

  unsigned long id =  (unsigned long int)threadIdx.x + blockIdx.x * blockDim.x;
  hipfftComplex c;  
  
  if( id < dataSize ) {
    c = (hipfftComplex) hipConjf( (hipComplex) dev_ydata[id] ) ;
    dev_zdata[id] = (hipfftComplex) ComplexMul( dev_xdata[id], c ) ;
    cuPrintf("id %d zdata %f +i%f\r", id, dev_zdata[id].x, dev_zdata[id].y) ;
  }
  
}

///////////////////////////////////////////////

int main(int argc, char *argv[]) {

  cudaPrintfInit();

  ///////////////////////////
  // Import Data
  ///////////////////////////

  printf("Import Data ") ;
  
  // float *data ;
  // data = (float *) malloc( (unsigned long) Nx * Ny * sizeof(float) * BATCH );
  
  // FILE *file;
  // file = fopen("../../MATLAB/sensorData.dat","rb") ;

  // int dum ;
  // dum = fread(&data[0], sizeof(float), (unsigned long) Nx*Ny*BATCH, file) ;
  // fclose(file) ;

  // for(int i=0;i<10;i++) 
  //   printf("%f %f | ", data[i], data[i+Nx] ) ;
  // printf("\n") ;

  // unsigned long Nfft = pow( 2, ( ceil( log2( (float) ( Nx+Ny-1 ) ) ) ) ) ; 

  // printf("Nx %lu Ny %lu Nfft %lu ", Nx, Ny, Nfft) ;

  // float *xdata,*ydata ;
  // xdata = (float *) malloc( (unsigned long) Nfft * sizeof(float) * BATCH );
  // ydata = (float *) malloc( (unsigned long) Nfft * sizeof(float) * BATCH );

  // for(unsigned long i=0;i<Nfft;i++) {

  //   if(i<Nx)
  //     // xdata[i] = cos( (float) i );
  //     xdata[i] = data[i] ;
  //   else
  //     xdata[i] = 0. ;

  //   if(i<Ny)
  //     // ydata[i] = sin( (float) i );
  //   ydata[i] = data[i+Nx] ; 
  //   else
  //     ydata[i] = 0. ;    
  // }

  unsigned long Nfft ; 
  float **SpkTimes ;
  ImportSpikeTrains(Nfft, SpkTimes) ;
  
  ///////////////////////////
  // cufft utils
  ///////////////////////////

  printf("... Done\nAllocate device pointers ") ;
  
  hipfftReal *dev_xdata, *dev_ydata, *dev_xyCorr ;
  hipfftComplex *dev_iXdata, *dev_iYdata, *dev_iZdata ;

  cudaCheck( hipMalloc((void**)&dev_xdata, (unsigned long) Nfft * sizeof(hipfftReal) * BATCH) ) ;
  cudaCheck( hipMalloc((void**)&dev_ydata, (unsigned long) Nfft * sizeof(hipfftReal) * BATCH) ) ;
  cudaCheck( hipMalloc((void**)&dev_xyCorr, (unsigned long) Nfft * sizeof(hipfftReal) * BATCH) ) ;

  cudaCheck( hipMalloc((void**)&dev_iXdata, (unsigned long) (Nfft/2+1) * sizeof(hipfftComplex) * BATCH) ) ;
  cudaCheck( hipMalloc((void**)&dev_iYdata, (unsigned long) (Nfft/2+1) * sizeof(hipfftComplex) * BATCH) ) ;
  cudaCheck( hipMalloc((void**)&dev_iZdata, (unsigned long) (Nfft/2+1) * sizeof(hipfftComplex) * BATCH) ) ;
  
  float *xyCorr= NULL ;
  xyCorr = (float *) malloc( (unsigned long) Nfft * sizeof(float) );
  
  printf("... Done\nCopy Host to Dev") ;
  
  cudaCheck( hipMemcpy(dev_xdata, xdata, (unsigned long) Nfft * sizeof(hipfftReal) * BATCH, hipMemcpyHostToDevice) ) ; 
  cudaCheck( hipMemcpy(dev_ydata, ydata, (unsigned long) Nfft * sizeof(hipfftReal)* BATCH, hipMemcpyHostToDevice) ) ; 

  ///////////////////////////
  
  hipfftHandle plan ;

  if (hipfftPlan1d(&plan, Nfft, HIPFFT_R2C, BATCH) != HIPFFT_SUCCESS){
    fprintf(stderr, "CUFFT error: Plan creation failed");
    exit(-1) ;
  }
  
  ///////////////////////////

  printf("... Done\nFirst FFT ") ;
  /* Use the CUFFT plan to transform the signal in place. */
  if (hipfftExecR2C(plan, dev_xdata, dev_iXdata) != HIPFFT_SUCCESS){
    fprintf(stderr, "CUFFT error: ExecR2C Forward failed");
    exit(-1) ;
  }

  printf("... Done\n Second FFT ") ;
  if (hipfftExecR2C(plan, dev_ydata, dev_iYdata) != HIPFFT_SUCCESS){
    fprintf(stderr, "CUFFT error: ExecR2C Forward failed\n");
    exit(-1) ;
  }
  
  if (hipDeviceSynchronize() != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to synchronize\n");
    exit(-1) ;
  }

  ///////////////////////////////////////
  // Kernel Setup for multiplication
  ///////////////////////////////////////

  printf("... Done\n  Multiply FFT ") ;

  ///////////
  // on CPU
  ///////////

  // hipfftComplex *iXdata, *iYdata, *iZdata ;
  // iXdata = (hipfftComplex *) malloc( (unsigned long) (Nfft/2+1) * sizeof(hipfftComplex) );
  // iYdata = (hipfftComplex *) malloc( (unsigned long) (Nfft/2+1) * sizeof(hipfftComplex) );
  // iZdata = (hipfftComplex *) malloc( (unsigned long) (Nfft/2+1) * sizeof(hipfftComplex) );

  // cudaCheck( hipMemcpy(iXdata, dev_iXdata, (unsigned long) (Nfft/2+1) * sizeof(hipfftComplex), hipMemcpyDeviceToHost) ) ; 
  // cudaCheck( hipMemcpy(iYdata, dev_iYdata, (unsigned long) (Nfft/2+1) * sizeof(hipfftComplex), hipMemcpyDeviceToHost) ) ; 

  // for(unsigned long i=0;i<Nfft/2+1;i++)
  //   iZdata[i] = (hipfftComplex) ComplexMul( iXdata[i], hipConjf( iYdata[i] ) ) ;
    
  // cudaCheck( hipMemcpy(dev_iZdata, iZdata, (unsigned long) (Nfft/2+1) * sizeof(hipfftComplex), hipMemcpyHostToDevice) ) ; 

  // for(int i=0;i<2;i++)
  //   printf("%f +i%f * %f +i%f = %f +i%f \n", iXdata[i].x, iXdata[i].y, iYdata[i].x, iYdata[i].y, iZdata[i].x, iZdata[i].y) ;

  ///////////
  // on GPU
  ///////////
  
  int ThreadsPerBlock = N_THREADS ;
  int BlocksPerGrid = ( Nfft + ThreadsPerBlock-1 ) / ThreadsPerBlock;

  if(BlocksPerGrid > 65536) {
    printf("BlocksPerGrid exceds valid number of allowed blocks of 65536");
    exit(-1);
  }

  cudaMultiply<<<BlocksPerGrid, ThreadsPerBlock>>>(dev_iXdata, dev_iYdata, dev_iZdata, (unsigned long) Nfft/2+1 ) ; 
  cudaPrintfDisplay(stdout, true);
  
  if (hipDeviceSynchronize() != hipSuccess){
    fprintf(stderr, "Cuda error: Failed to synchronize\n");
    exit(-1) ;
  }

  ///////////////////////////

  printf("... Done\n   Invert FFT ") ;
  
  if (hipfftPlan1d(&plan, Nfft, HIPFFT_C2R, BATCH) != HIPFFT_SUCCESS){
    fprintf(stderr, "CUFFT error: Plan creation failed");
    exit(-1);
  }

  if (hipfftExecC2R(plan, dev_iZdata, dev_xyCorr) != HIPFFT_SUCCESS){
    fprintf(stderr, "CUFFT error: ExecC2R Forward failed\n");
    exit(-1) ;
  }

  if (hipDeviceSynchronize() != hipSuccess){
    fprintf(stderr, "Cuda error: Failed to synchronize\n");
    exit(-1) ;
  }

  hipfftDestroy(plan) ;

  ///////////////////////////
  
  printf("... Done\nFree Memory \n") ;
  
  free(data) ;
  free(xdata) ;
  free(ydata) ;
  
  hipFree(dev_xdata) ; 
  hipFree(dev_ydata) ;

  hipFree(dev_iXdata) ;
  hipFree(dev_iYdata) ;
  hipFree(dev_iZdata) ;

  ///////////////////////////
  
  cudaCheck( hipMemcpy(xyCorr, dev_xyCorr, (unsigned long) Nfft * sizeof(float), hipMemcpyDeviceToHost) ) ; 
  
  float xNorm, yNorm ;
  xNorm = Norm(xdata,Nfft) ;
  yNorm = Norm(ydata,Nfft) ;

  for(unsigned long i=0;i<Nfft;i++)
    xyCorr[i] = xyCorr[i] /xNorm /yNorm /(float)Nfft;

  file = fopen("xyCorr.dat","wb");
  fwrite(xyCorr, sizeof(float), Nfft, file) ;
  fclose(file) ;

  ///////////////////////////
  
  hipFree(dev_xyCorr) ;
  hipHostFree(xyCorr) ;

  ///////////////////////////
  
  return 0 ;
}

