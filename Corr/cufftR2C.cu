#define NX 256
#define BATCH 1

hipfftHandle plan;
hipfftComplex *data;

hipMalloc((void**)&data, sizeof(hipfftComplex)*(NX/2+1)*BATCH);

if (hipGetLastError() != hipSuccess){
  fprintf(stderr, "Cuda error: Failed to allocate\n");
  return;
 }

if (hipfftPlan1d(&plan, NX, HIPFFT_R2C, BATCH) != HIPFFT_SUCCESS){
  fprintf(stderr, "CUFFT error: Plan creation failed");
  return;
 }

/* Use the CUFFT plan to transform the signal in place. */
if (hipfftExecR2C(plan, (hipfftReal*)data, data) != HIPFFT_SUCCESS){
  fprintf(stderr, "CUFFT error: ExecR2C Forward failed");
  return;
 }

if (hipDeviceSynchronize() != hipSuccess){
  fprintf(stderr, "Cuda error: Failed to synchronize\n");
  return;
 }

hipfftDestroy(plan);
hipFree(data);